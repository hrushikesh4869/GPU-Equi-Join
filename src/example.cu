#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <thread>

using namespace std;

__global__ void write_output(int* input, int *output, int* buffer1, int* index, int* current_buffer, int* count, int *isfull, int bufferSize, int dataSize){
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("inside the kernel\n");
    if(threadid<dataSize){

        int idx = atomicAdd(index,1);

        if(idx < bufferSize/2)
        {
            buffer1[idx + *current_buffer] = input[(*count)*bufferSize/2 + idx + *current_buffer];
            printf("inside the kernel %d\n",idx);
        }
        else if(idx == bufferSize/2)
        {   
            *isfull = 1;

            (*count)++;

            if(*current_buffer != 0)
            {
                *current_buffer = 0;
            }
            else
            {
                *current_buffer = bufferSize/2;
            }
            idx = 0;
            *index = 1;

            buffer1[idx + *current_buffer] = input[(*count)*bufferSize/2 + idx + *current_buffer];
        }
        else
        {
            while(*index >= bufferSize/2)
            {
                ;
            }

            idx = atomicAdd(index,1);
            buffer1[idx + *current_buffer] = input[(*count)*bufferSize/2 + idx + *current_buffer];
        }
    }

}

void copy_thread(int *buffer1, int* output, int bufferSize, int *isfull)
{
    int count = 0;
    int current_buffer = 0;

    // create cuda stream to copy data from buffer to output
    hipStream_t stream;
    hipStreamCreate(&stream);
    //cout<<"created thread \n";
    while(true)
    {
        if(*isfull == 1)
        {
            cout<<"inside the thread\n";
            hipMemcpyAsync(output + (count)*bufferSize/2, buffer1 + current_buffer, (bufferSize/2)*sizeof(int), hipMemcpyDeviceToHost,stream);
            hipDeviceSynchronize();
            cout<<"inside the thread 22\n";
            
            if(current_buffer == 0)
            {
                current_buffer = bufferSize/2;
            }
            else
            {
                current_buffer = 0;
            }
            *isfull = 0;
            count++;
            
        }
    }
}


int main(){
    int dataSize = 1e3;
    int bufferSize = 1e2;

    int *buffer1,*buffer2,*offset,*currentOffset,*inputDev,*count, *isfull;

    int *input,*output;

    input = (int*)malloc(dataSize*sizeof(int));
    output = (int*)malloc(dataSize*sizeof(int));

    for(int i = 0; i<dataSize; i++){
        input[i] = i;
    }

    hipMalloc(&buffer1,bufferSize*sizeof(int));
    hipMalloc(&buffer2,bufferSize*sizeof(int));
    hipHostAlloc(&isfull,sizeof(int),hipHostMallocDefault);

    hipMalloc(&offset,sizeof(int));
    hipMalloc(&count,sizeof(int));
    hipMalloc(&currentOffset,sizeof(int));

    hipMalloc(&inputDev,dataSize*sizeof(int));

    hipMemcpy(inputDev,input,dataSize*sizeof(int),hipMemcpyHostToDevice);
    
    hipMemset(offset,0,sizeof(int));
    hipMemset(count,0,sizeof(int));
    hipMemset(currentOffset,0,sizeof(int));
    hipMemset(isfull,0,sizeof(int));
    hipMemset(buffer1,-1,bufferSize*sizeof(int));

    // spawn a thread to copy data from buffer to output
    cout<<*isfull<<endl;
    thread t1(copy_thread,buffer1,output,bufferSize,isfull);
    t1.detach();
    
    write_output<<<dataSize/32+1,32>>>(inputDev,output,buffer1,offset,currentOffset,count,isfull,bufferSize,dataSize);
    // print cuda last error
    cout<<hipGetLastError()<<endl;
    hipDeviceSynchronize();

    for(int i = 0; i<dataSize; i++){
        cout<<output[i]<<" ";
    }

    hipFree(buffer1);
    hipFree(buffer2);
    hipFree(offset);
    hipFree(count);
    hipFree(currentOffset);
    hipFree(inputDev);
    hipHostFree(isfull);
    free(input);
    free(output);
}