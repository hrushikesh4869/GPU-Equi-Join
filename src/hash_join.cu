#include "hip/hip_runtime.h"
#include "string_hash.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include "linearprobing.h"
 
using namespace std;
 

__device__ __forceinline__  uint32_t hash_re(uint32_t k) {
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (kHashTableCapacity - 1);
}
 
void hash_join_util(vector<Tuple> &data,vector<KeyValue> &hashedData, string file) {
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
    start = std::chrono::high_resolution_clock::now();
    load_data(data, file);
    end = std::chrono::high_resolution_clock::now();
    cout << "Time taken to load data: "<< std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count()<< " nanoseconds" << endl;
 
    // count time taken to generate hash
 
    start = std::chrono::high_resolution_clock::now();
    generate_hash(data, hashedData);
    end = std::chrono::high_resolution_clock::now();
    cout << "Time taken to generate hash: "<< std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count()*1e-6<< " milliseconds" << endl;
}
 

__global__ void compute_upperbound(KeyValue *hashTable,KeyValue* hashedDataS, uint32_t sizeS, uint32_t sizeR, int *upperbound){
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
 
    if(threadid< sizeS){
        uint32_t key = hashedDataS[threadid].key;
        uint32_t slot;
        
        slot = hash_re(key);
        while (true) {
            
            if (hashTable[slot].key == key) {
                int idx = atomicAdd(upperbound,1);
            }
            if (hashTable[slot].key == kEmpty) {
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}

__global__ void gpu_hash_join(KeyValue *hashTable,KeyValue* hashedDataS, uint32_t sizeS, uint32_t sizeR, Result *result ,int *count){
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
 
    if(threadid< sizeS){
        uint32_t key = hashedDataS[threadid].key;
        uint32_t slot;
        
        slot = hash_re(key);
        while (true) {
            
            if (hashTable[slot].key == key) {
                int idx = atomicAdd(count,1);
                result[idx].rid = hashTable[slot].value;
                result[idx].sid = hashedDataS[threadid].value;
            }
            if (hashTable[slot].key == kEmpty) {
                return;
            }
            slot = (slot + 1) & (kHashTableCapacity - 1);
        }
    }
}
 
int main() {
 
    vector<Tuple> dataR, dataS;
    vector<KeyValue> hashedDataR, hashedDataS;
    int *count, sizeResult = 0, numResults = 0;
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
    
    string dir = "/home/naruto/Documents/DBMS_Project/data/";
 
    hash_join_util(dataR, hashedDataR,dir+"table_r3.csv");
    hash_join_util(dataS, hashedDataS,dir+"table_s3.csv");
 
    int sizeR = hashedDataR.size();
    int sizeS = hashedDataS.size();
    
    start = std::chrono::high_resolution_clock::now();
    KeyValue* hashedDataRDev = &hashedDataR[0];
    KeyValue* pHashTable = create_hashtable();
    insert_hashtable(pHashTable, hashedDataRDev, sizeR);
    
 
    KeyValue* hashedDataSDev;
 
    hipMalloc(&hashedDataSDev, sizeof(KeyValue) * sizeS);
    hipMemcpy(hashedDataSDev, hashedDataS.data(), sizeof(KeyValue) * sizeS, hipMemcpyHostToDevice);
    

    
    int *upperbound;
    hipMalloc(&upperbound,sizeof(int));
    hipMemset(upperbound,0,sizeof(int));
    
    compute_upperbound<<<max(sizeS/32,1), 32>>>(pHashTable, hashedDataSDev, sizeS, sizeR, upperbound);
    
    hipMemcpy(&sizeResult, upperbound, sizeof(int), hipMemcpyDeviceToHost);
    
    cout<<sizeResult<<endl;

    Result *result = (Result*) malloc(sizeof(Result) * sizeResult);
    Result* resultDev;

    hipMalloc(&resultDev, sizeof(Result) * sizeResult);
    hipMemset(resultDev, -1, sizeof(Result) * sizeResult);
     
    hipMalloc(&count,sizeof(int));
 
    hipMemset(count,0,sizeof(int));
    gpu_hash_join<<<max(sizeS/32,1), 32>>>(pHashTable, hashedDataSDev, sizeS, sizeR, resultDev, count);

    cout<<hipGetErrorString(hipGetLastError())<<endl;
    hipMemcpy(result, resultDev, sizeof(Result) * sizeResult, hipMemcpyDeviceToHost);

    int total=0;
    hipMemcpy(&total, count, sizeof(int), hipMemcpyDeviceToHost);

    end = std::chrono::high_resolution_clock::now();
 
    cout << "Time taken to do the join: "<< std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count()*1e-6<< " milliseconds" << endl;
    
    
    cout<<total<<endl;
 
    start = std::chrono::high_resolution_clock::now();
    
    materialize_results(dataR, dataS,result, sizeResult,total);
 
    end = std::chrono::high_resolution_clock::now();
 
    cout << "Time taken to materialize the join: "<< std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count()*1e-6<< " milliseconds" << endl;
 
    hipFree(count);
    hipHostFree(resultDev);
    hipFree(hashedDataSDev);
 
  return 0;
}